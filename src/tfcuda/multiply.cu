#include "hip/hip_runtime.h"
//
// Created by connor on 8/9/21.
//


#include <hip/hip_runtime.h> // hipMalloc, hipMemcpy, etc.
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE
#include "multiply.h"
#include <iostream>
#include "chrono"

#define BLOCK_SIZE 32



// TODO: make array multiply
__global__ void cuda_hello(DTYPE *a, DTYPE *b, DTYPE *c) {
    printf("hello cuda \n");
    *c = *a * *b;
}

__global__ void matrixMultiplicationKernel(DTYPE *a, DTYPE *b, DTYPE *c, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    DTYPE sum = 0;
    if( col < N && row < N)
    {
        for(int i = 0; i < N; i++)
        {
            sum += a[row * N + i] * b[i * N + col];
        }
        c[row * N + col] = sum;
    }
}

void matrixMultiplicationCPU(DTYPE *a, DTYPE *b, DTYPE *c, int N){
    DTYPE sum;
    for (int row=0; row<N; row++){
        for (int col=0; col<N; col++){
            sum = 0.f;
            for (int n=0; n<N; n++){
                sum += a[row*N+n]*b[n*N+col];
            }
            c[row*N+col] = sum;
        }
    }

}

void matrixMultiplication(DTYPE *a, DTYPE *b, DTYPE *d_c, int N, bool timeFunc){
    hipError_t status;
    DTYPE *d_a, *d_b;
    long size = (long) N * (long) N;

    status = hipMalloc((void **)&d_a, size * sizeof(DTYPE));
    CUDAMALLOCCHECK(d_a, size, DTYPE, status);
    long t = size * sizeof(DTYPE);
    status = hipMalloc((void **)&d_b, size * sizeof(DTYPE));
    CUDAMALLOCCHECK(d_b, size, DTYPE, status);

    status = hipMemcpy(d_a, a, size * sizeof(DTYPE), hipMemcpyHostToDevice);
    CUDAMEMCPYCHECK(d_a, size, DTYPE, status);
    status = hipMemcpy(d_b, b, size * sizeof(DTYPE), hipMemcpyHostToDevice);
    CUDAMEMCPYCHECK(d_b, size, DTYPE, status);

//    dim3 threadsPerBlock(N, N);
//    dim3 blocksPerGrid(1, 1);
//    if (N*N > 512){
//        threadsPerBlock.x = 512;
//        threadsPerBlock.y = 512;
//        blocksPerGrid.x = ceil(double(N)/double(threadsPerBlock.x));
//        blocksPerGrid.y = ceil(double(N)/double(threadsPerBlock.y));
//    }
    unsigned int grid_rows = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    status = hipGetLastError () ; // clear error status

    std::chrono::steady_clock::time_point begin = std::chrono::steady_clock::now();
    matrixMultiplicationKernel<<<dimGrid,dimBlock>>>(d_b, d_a, d_c, N);

    hipDeviceSynchronize () ;

    if (timeFunc) {
        std::chrono::steady_clock::time_point end = std::chrono::steady_clock::now();
        std::cout << "CUDA Mat Multiply call took = "
                  << std::chrono::duration_cast<std::chrono::seconds>(end - begin).count() << "[s]" << std::endl;
    }

    status = hipGetLastError();        // Get error code

    if ( status != hipSuccess )
    {
        printf("CUDA Error: %s\n", hipGetErrorString(status));
        exit(-1);
    }
//    hipFree(d_a);
//    hipFree(d_b);
}


__global__ void cudaCopyKernel(DTYPE * des, DTYPE * src, const long N){
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    if(tx<N)
        des[tx]=src[tx];
}

void cudaCopy(DTYPE *des, DTYPE* src, long N){
    unsigned int grid_cols = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, 1);
    dim3 dimBlock(BLOCK_SIZE, 1);

    cudaCopyKernel<<<dimGrid,dimBlock>>>(des, src, N);
    hipDeviceSynchronize();

}

void multiplyPointer(DTYPE *d_a, DTYPE *d_b, DTYPE *d_c){
    cuda_hello<<<1,1>>>(d_a, d_b, d_c);
}

void multiplyTensor(DTYPE *d_a, DTYPE *d_b, DTYPE *d_c){
    cuda_hello<<<1,1>>>(d_a, d_b, d_c);
}


int multiply(DTYPE a, DTYPE b){
    DTYPE c;
    DTYPE *d_a, *d_b, *d_c;
    DTYPE size = sizeof(DTYPE);

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
    //    hipMemcpy(d_c, &c, size, hipMemcpyHostToDevice);

    cuda_hello<<<1,1>>>(d_a, d_b, d_c);

    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    printf("result is %f \n", c);
    return c;
}
