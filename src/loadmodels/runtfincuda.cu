//
// Created by connor on 8/9/21.
//

#include <hip/hip_runtime.h> // cudaMalloc, cudaMemcpy, etc.
#include <stdio.h>            // printf
#include <stdlib.h>           // EXIT_FAILURE

__global__ void cudaCopy(double* des, double* sour, const int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if( i <  N) {
        des[i] = sour[i];
    }

}

__global__ void cuda_hello(int *a, int *b, int *c) {
    *c = *a + *b;
}

int main() {
    int a, b, c;
    int *d_a, *d_b, *d_c;
    int size = sizeof(int);

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    a = 10;
    b = 7;

    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
    //    cudaMemcpy(d_c, &c, size, cudaMemcpyHostToDevice);



    cuda_hello<<<1,1>>>(d_a, d_b, d_c);

    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    printf("Hello World from GPU! Value is %d\n", c);

    return 0;
}
